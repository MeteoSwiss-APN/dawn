#include "hip/hip_runtime.h"
#define DAWN_GENERATED 1
#undef DAWN_BACKEND_T
#define DAWN_BACKEND_T CUDA
#ifndef BOOST_RESULT_OF_USE_TR1
 #define BOOST_RESULT_OF_USE_TR1 1
#endif
#ifndef BOOST_NO_CXX11_DECLTYPE
 #define BOOST_NO_CXX11_DECLTYPE 1
#endif
#ifndef GRIDTOOLS_DAWN_HALO_EXTENT
 #define GRIDTOOLS_DAWN_HALO_EXTENT 0
#endif
#ifndef BOOST_PP_VARIADICS
 #define BOOST_PP_VARIADICS 1
#endif
#ifndef BOOST_FUSION_DONT_USE_PREPROCESSED_FILES
 #define BOOST_FUSION_DONT_USE_PREPROCESSED_FILES 1
#endif
#ifndef BOOST_MPL_CFG_NO_PREPROCESSED_HEADERS
 #define BOOST_MPL_CFG_NO_PREPROCESSED_HEADERS 1
#endif
#ifndef GT_VECTOR_LIMIT_SIZE
 #define GT_VECTOR_LIMIT_SIZE 30
#endif
#ifndef BOOST_FUSION_INVOKE_MAX_ARITY
 #define BOOST_FUSION_INVOKE_MAX_ARITY GT_VECTOR_LIMIT_SIZE
#endif
#ifndef FUSION_MAX_VECTOR_SIZE
 #define FUSION_MAX_VECTOR_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#ifndef FUSION_MAX_MAP_SIZE
 #define FUSION_MAX_MAP_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#ifndef BOOST_MPL_LIMIT_VECTOR_SIZE
 #define BOOST_MPL_LIMIT_VECTOR_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#include <driver-includes/gridtools_includes.hpp>
using namespace gridtools::dawn;
namespace dawn_generated{
namespace cuda{
__global__ void __launch_bounds__(128)  generated_stencil59_ms58_kernel(const int isize, const int jsize, const int ksize, const int stride_111_1, const int stride_111_2, ::dawn::float_type * const in, ::dawn::float_type * const out) {

  // Start kernel
  const unsigned int nx = isize;
  const unsigned int ny = jsize;
  const int block_size_i = (blockIdx.x + 1) * 32 < nx ? 32 : nx - blockIdx.x * 32;
  const int block_size_j = (blockIdx.y + 1) * 4 < ny ? 4 : ny - blockIdx.y * 4;

  // computing the global position in the physical domain

  // In a typical cuda block we have the following regions

  // aa bbbbbbbb cc

  // aa bbbbbbbb cc

  // hh dddddddd ii

  // hh dddddddd ii

  // hh dddddddd ii

  // hh dddddddd ii

  // ee ffffffff gg

  // ee ffffffff gg

  // Regions b,d,f have warp (or multiple of warp size)

  // Size of regions a, c, h, i, e, g are determined by max_extent_t

  // Regions b,d,f are easily executed by dedicated warps (one warp for each line)

  // Regions (a,h,e) and (c,i,g) are executed by two specialized warp
  int iblock = 0 - 1;
  int jblock = 0 - 1;
if(threadIdx.y < +4) {
    iblock = threadIdx.x;
    jblock = (int)threadIdx.y + 0;
}
  // initialized iterators
  int idx111 = (blockIdx.x*32+iblock)*1+(blockIdx.y*4+jblock)*stride_111_1;

  // Pre-fill of kcaches
for(int k = 0+0; k <= 10+0; ++k) {

    // Head fill of kcaches
  if(iblock >= 0 && iblock <= block_size_i -1 + 0 && jblock >= 0 && jblock <= block_size_j -1 + 0) {
::dawn::float_type dx;
{
  out[idx111] = (((int) -4 * (__ldg(&(in[idx111])) + (__ldg(&(in[idx111+1*1])) + (__ldg(&(in[idx111+1*-1])) + (__ldg(&(in[idx111+stride_111_1*-1])) + __ldg(&(in[idx111+stride_111_1*1]))))))) / (dx * dx));
}
  }
    // Flush of kcaches

    // Flush of kcaches

    // Slide kcaches

    // increment iterators
    idx111+=stride_111_2;
}
  // Final flush of kcaches

  // Final flush of kcaches

  // Final flush of kcaches

  // jump iterators to match the beginning of next interval
  idx111 += stride_111_2*(4);

  // Pre-fill of kcaches
for(int k = 15+0; k <=  ksize - 1 + 0+0; ++k) {

    // Head fill of kcaches
  if(iblock >= 0 && iblock <= block_size_i -1 + 0 && jblock >= 0 && jblock <= block_size_j -1 + 0) {
{
  out[idx111] = (int) 10;
}
  }
    // Flush of kcaches

    // Flush of kcaches

    // Slide kcaches

    // increment iterators
    idx111+=stride_111_2;
}
  // Final flush of kcaches

  // Final flush of kcaches

  // Final flush of kcaches
}

class generated {
public:

  struct sbase : public timer_cuda {

    sbase(std::string name) : timer_cuda(name){}

    double get_time() {
      return total_time();
    }
  };

  struct stencil_59 : public sbase {

    // Members

    // Temporary storage typedefs
    using tmp_halo_t = gridtools::halo< 0,0, 0, 0, 0>;
    using tmp_meta_data_t = storage_traits_t::storage_info_t< 0, 5, tmp_halo_t >;
    using tmp_storage_t = storage_traits_t::data_store_t< ::dawn::float_type, tmp_meta_data_t>;
    const gridtools::dawn::domain m_dom;
  public:

    stencil_59(const gridtools::dawn::domain& dom_, int rank, int xcols, int ycols) : sbase("stencil_59"), m_dom(dom_){}
    static constexpr dawn::driver::cartesian_extent in_extent = {-1,1, -1,1, 0,0};
    static constexpr dawn::driver::cartesian_extent out_extent = {0,0, 0,0, 0,0};

    void run(storage_ijk_t in_ds, storage_ijk_t out_ds) {

      // starting timers
      start();
      {;
      gridtools::data_view<storage_ijk_t> in= gridtools::make_device_view(in_ds);
      gridtools::data_view<storage_ijk_t> out= gridtools::make_device_view(out_ds);
      const unsigned int nx = m_dom.isize() - m_dom.iminus() - m_dom.iplus();
      const unsigned int ny = m_dom.jsize() - m_dom.jminus() - m_dom.jplus();
      const unsigned int nz = m_dom.ksize() - m_dom.kminus() - m_dom.kplus();
      dim3 threads(32,4+0,1);
      const unsigned int nbx = (nx + 32 - 1) / 32;
      const unsigned int nby = (ny + 4 - 1) / 4;
      const unsigned int nbz = 1;
      dim3 blocks(nbx, nby, nbz);
      generated_stencil59_ms58_kernel<<<blocks, threads>>>(nx,ny,nz,in_ds.strides()[1],in_ds.strides()[2],(in.data()+in_ds.get_storage_info_ptr()->index(in.begin<0>(), in.begin<1>(),0 )),(out.data()+out_ds.get_storage_info_ptr()->index(out.begin<0>(), out.begin<1>(),0 )));
      };

      // stopping timers
      pause();
    }
  };
  static constexpr const char* s_name = "generated";
  stencil_59 m_stencil_59;
public:

  generated(const generated&) = delete;

  // Members

  // Stencil-Data

  generated(const gridtools::dawn::domain& dom, int rank = 1, int xcols = 1, int ycols = 1) : m_stencil_59(dom, rank, xcols, ycols){}

  template<typename S>
  void sync_storages(S field) {
    field.sync();
  }

  template<typename S0, typename ... S>
  void sync_storages(S0 f0, S... fields) {
    f0.sync();
    sync_storages(fields...);
  }

  void run(storage_ijk_t in, storage_ijk_t out) {
    sync_storages(in,out);
    m_stencil_59.run(in,out);
;
    sync_storages(in,out);
  }

  std::string get_name()  const {
    return std::string(s_name);
  }

  void reset_meters() {
m_stencil_59.reset();  }

  double get_total_time() {
    double res = 0;
    res +=m_stencil_59.get_time();
    return res;
  }
};
} // namespace cuda
} // namespace dawn_generated
